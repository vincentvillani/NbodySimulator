#include "hip/hip_runtime.h"
/*
 * Simulation.cu
 *
 *  Created on: 6 Aug 2016
 *      Author: vincent
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <stdint.h>

#include "../Header/Simulation.h"

#include "../Header/Device_Particles.h"
#include "../Header/Host_Particles.h"
#include "../Header/ParticlesUtility.h"
#include "../Header/kernels.h"


#define BLOCK_DIM 1024

void DeviceSimulation(Host_Particles* hostParticles, Device_Particles* deviceParticles, float* d_timeDelta, float* d_mass)
{
	dim3 block(BLOCK_DIM);
	dim3 grid( ceilf( hostParticles->h_particleNumber / (float)BLOCK_DIM) );
	//printf("block: %u\ngrid: %u\n", block.x, grid.x);

	CalculateForcesGlobal<<<grid, block>>> (deviceParticles->d_positions, deviceParticles->d_velocities, deviceParticles->d_particleNumber, d_timeDelta, d_mass);
	//gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( hipDeviceSynchronize() );

	UpdatePositionsGlobal<<<grid, block>>>(deviceParticles->d_positions, deviceParticles->d_velocities, deviceParticles->d_particleNumber, d_timeDelta);
	//gpuErrchk( hipPeekAtLastError() );
	//gpuErrchk( hipDeviceSynchronize() );
}


void WriteHostDataToFile(Host_Particles* hostParticles, uint64_t frameNumber, FILE* outputFile)
{

	fwrite(&frameNumber, sizeof(uint64_t), 1, outputFile);
	fwrite(hostParticles->h_positions, sizeof(float) * hostParticles->h_arrayLength, 1, outputFile);

}


void WriteHeaderInformationToOutputFile(Host_Particles* hostParticles, uint64_t h_totalNumberOfSteps, float h_timeDelta, float mass, FILE* outputFile)
{
	uint64_t headerByteSize = sizeof(uint64_t) + sizeof(uint64_t) + sizeof(uint64_t) + sizeof(float) + sizeof(float);
	uint64_t expectedFileByteSize = headerByteSize + (h_totalNumberOfSteps * hostParticles->h_particleNumber * 3 * sizeof(float)) +
			h_totalNumberOfSteps * sizeof(uint64_t);

	fwrite(&expectedFileByteSize, sizeof(uint64_t),1, outputFile);
	fwrite(&h_totalNumberOfSteps, sizeof(uint64_t), 1, outputFile);
	fwrite(&(hostParticles->h_particleNumber), sizeof(uint64_t), 1, outputFile);
	fwrite(&h_timeDelta, sizeof(float), 1, outputFile);
	fwrite(&mass, sizeof(float), 1, outputFile);

}


void Simulate(Host_Particles* hostParticles, Device_Particles* deviceParticles, uint64_t h_totalNumberOfSteps, float h_timeDelta, float h_mass)
{

	//Open up the output file
	FILE* outputFile = fopen("OutputFile.sim", "wb");

	if(outputFile == NULL)
	{
		fprintf(stderr, "Simulate: Unable to open output file\n");
		exit(1);
	}

	//Write the file header
	WriteHeaderInformationToOutputFile(hostParticles, h_totalNumberOfSteps, h_timeDelta, h_mass, outputFile);


	//Allocate memory for the time delta and mass and copy it to the device
	float* d_timeDelta = NULL;
	hipMalloc(&d_timeDelta, sizeof(float));
	hipMemcpy(d_timeDelta, &h_timeDelta, sizeof(float), hipMemcpyHostToDevice);

	float* d_mass = NULL;
	hipMalloc(&d_mass, sizeof(float));
	hipMemcpy(d_mass, &h_mass, sizeof(float), hipMemcpyHostToDevice);



	for(uint64_t i = 0; i < h_totalNumberOfSteps; ++i)
	{
		//Sync with the device
		if( hipDeviceSynchronize() != hipSuccess)
		{
			fprintf(stderr, "Simulate: hipError_t when trying to sync\n");
			exit(1);
		}

		//Copy the new position data over to the host
		CopyDeviceParticlesPositionsToHost(deviceParticles, hostParticles);

		//Call the two simulation kernels, one by one (async)
		DeviceSimulation(hostParticles, deviceParticles, d_timeDelta, d_mass);

		//While the simulation is occuring, write the host data to a file
		WriteHostDataToFile(hostParticles, i, outputFile);

		printf("Frame: %llu/%llu\n", i, h_totalNumberOfSteps);

	}


	fclose(outputFile);

}



