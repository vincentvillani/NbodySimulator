/*
 * Device_Particles.cu
 *
 *  Created on: 6 Aug 2016
 *      Author: vincent
 */


#include "../Header/Device_Particles.h"

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

Device_Particles::Device_Particles(uint64_t particleNum)
{

	d_particleNumber = NULL;
	d_arrayLength = NULL;
	d_positions = NULL;
	d_velocities = NULL;


	size_t arrayByteSize = sizeof(float) * particleNum * 3;

	hipError_t hipError_t;

	//Allocate memory
	hipError_t = hipMalloc(&d_particleNumber, sizeof(uint64_t));
	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_Particles::Device_Particles: Allocating memory for d_particleNumber failed\n");
		exit(1);
	}

	hipError_t = hipMalloc(&d_arrayLength, sizeof(uint64_t));
	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_Particles::Device_Particles: Allocating memory for d_arrayLength failed\n");
		exit(1);
	}

	hipError_t = hipMalloc(&d_positions, arrayByteSize);
	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_Particles::Device_Particles: Allocating memory for d_positions failed\n");
		exit(1);
	}

	hipError_t = hipMalloc(&d_velocities, arrayByteSize);
	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "Device_Particles::Device_Particles: Allocating memory for d_positions failed\n");
		exit(1);
	}

	//Copy the particle number and length values over
	uint64_t h_arrayLength = particleNum * 3;
	hipMemcpy(d_particleNumber, &particleNum, sizeof(uint64_t), hipMemcpyHostToDevice);
	hipMemcpy(d_arrayLength, &h_arrayLength, sizeof(uint64_t), hipMemcpyHostToDevice);

}

Device_Particles::~Device_Particles()
{
	if(d_particleNumber != NULL)
		hipFree(d_particleNumber);

	if(d_arrayLength != NULL)
		hipFree(d_arrayLength);

	if(d_positions != NULL)
		hipFree(d_positions);

	if(d_velocities != NULL)
		hipFree(d_velocities);
}


