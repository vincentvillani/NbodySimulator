/*
 * main.cu
 *
 *  Created on: 6 Aug 2016
 *      Author: vincent
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "Header/Host_Particles.h"
#include "Header/Device_Particles.h"
#include "Header/ParticlesUtility.h"
#include "Header/Simulation.h"

#define PARTICLE_NUM 1000

int main()
{

	//Allocate memory for particles on the host
	Host_Particles hostParticles(PARTICLE_NUM);

	//Allocate memory for the particles on the device
	Device_Particles deviceParticles(PARTICLE_NUM);

	//Set the intial state on the host
	SetInitialParticleStateHost(&hostParticles);

	//Copy the initial state over to the device
	CopyHostParticlesToDevice(&hostParticles, &deviceParticles);

	//Run the simulation
	Simulate(&hostParticles, &deviceParticles, 180, 1.0f/60.0f);


	printf("Done...\n");

	return 0;
}



